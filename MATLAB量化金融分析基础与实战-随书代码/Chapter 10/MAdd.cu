#include "hip/hip_runtime.h"
definition __global__ void MAdd(float X[M][N], float Y[M][N], float Z[M][N]) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
int j = blockIdx.y * blockDim.y + threadIdx.y;
if (i < M && j < N) C[i][j] = A[i][j] + B[i][j];
 }
int main() { 
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(M / threadsPerBlock.x, N / threadsPerBlock.y); 
	MAdd<<<numBlocks, threadsPerBlock>>>(A, B, C); 
	return 0;
}